#include  "cuda_typedef.hpp"
#include  "cuda_error_check.cuh"
#include  "cuda_util.cuh"
#include  "GpuParticle2DContainer.hpp"
#include  <cuda_gl_interop.h>

GpuParticle2DContainer::GpuParticle2DContainer(int nParticle)
    : nParticle_(nParticle),
      nbytes_(nParticle * 2 * sizeof(float)),
      particles_(cuda::make_unique<float[]>(this->nbytes_))
{
}

GpuParticle2DContainer::~GpuParticle2DContainer()
{
    if (this->graphicsResource_ != nullptr)
    {
        this->unmapGraphicsResource();
    }

    ::hipDeviceReset();
}

void GpuParticle2DContainer::copyToDevice(float* hostMemoryPtr, size_t nbytes)
{
    ThrowInvalidArgumentExceptionIfTooLargeMemorySize(nbytes);

    float* deviceMemoryPtr = this->particles_.get();

    CHECK_CUDA_ERROR(::hipMemcpy(deviceMemoryPtr, hostMemoryPtr, nbytes, ::hipMemcpyHostToDevice));
}

void GpuParticle2DContainer::copyToHost(float* hostMemoryPtr, size_t nbytes)
{
    ThrowInvalidArgumentExceptionIfTooLargeMemorySize(nbytes);

    float* deviceMemoryPtr = this->particles_.get();

    CHECK_CUDA_ERROR(::hipMemcpy(hostMemoryPtr, deviceMemoryPtr, nbytes, ::hipMemcpyDeviceToHost));
}

void GpuParticle2DContainer::registerGraphicsResource(const GLuint& vbo)
{
    this->ThrowRuntimeErrorIfGraphicsResourceAlreadyRegistered();

    this->vbo_ = vbo;

    auto rawGraphicsResource = &this->graphicsResource_;
    CHECK_CUDA_ERROR(::hipGraphicsGLRegisterBuffer(rawGraphicsResource, vbo, hipGraphicsRegisterFlagsNone));
}

void GpuParticle2DContainer::unregisterGraphicsResource(void)
{
    this->ThrowRuntimeErrorIfGraphicsResourceNotRegistered();

    CHECK_CUDA_ERROR(::hipGraphicsUnregisterResource(this->graphicsResource_));

    this->graphicsResource_ = nullptr;
}

void GpuParticle2DContainer::mapGraphicsResource(void)
{
    this->ThrowRuntimeErrorIfGraphicsResourceNotRegistered();

    CHECK_CUDA_ERROR(::hipGraphicsMapResources(1, &this->graphicsResource_, 0));

    auto rawDeviceMemoryPtr = this->particles_.get();
    CHECK_CUDA_ERROR(::hipGraphicsResourceGetMappedPointer(reinterpret_cast<void **>(&rawDeviceMemoryPtr), nullptr, this->graphicsResource_));
}

void GpuParticle2DContainer::unmapGraphicsResource(void)
{
    this->ThrowRuntimeErrorIfGraphicsResourceNotRegistered();

    CHECK_CUDA_ERROR(::hipGraphicsUnmapResources(1, &this->graphicsResource_, 0));
}

